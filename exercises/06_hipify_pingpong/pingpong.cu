
#include <hip/hip_runtime.h>
#include <stdio.h>

/* Macro for checking GPU API return values */
#define gpuCheck(call)                                                                           \
do{                                                                                              \
    hipError_t gpuErr = call;                                                                   \
    if(hipSuccess != gpuErr){                                                                   \
        printf("GPU API Error - %s:%d: '%s'\n", __FILE__, __LINE__, hipGetErrorString(gpuErr)); \
        exit(1);                                                                                 \
    }                                                                                            \
}while(0)

void host_device_transfer(const char* direction){

    int loop_count = 50;

    for(int i=10; i<=27; i++){

        long int N = 1 << i;

        size_t bytes = N * sizeof(double);

        float milliseconds = 0.0;

        double *h_A;
        gpuCheck( hipHostMalloc(&h_A, bytes, hipHostMallocDefault) );

        double *d_A;
        gpuCheck( hipMalloc(&d_A, bytes) );

        hipEvent_t start, stop;
        gpuCheck( hipEventCreate(&start) );
        gpuCheck( hipEventCreate(&stop) );

        for(int j=0; j<N; j++){
            h_A[j] = (double)rand()/(double)RAND_MAX;
        }

        /* Warm-up loop */
        if( strcmp(direction, "H2D") == 0){

            for(int iteration=0; iteration<5; iteration++){
                gpuCheck( hipMemcpy(d_A, h_A, bytes, hipMemcpyHostToDevice) );
            }

        }
        else if( strcmp(direction, "D2H") == 0){

            for(int iteration=0; iteration<5; iteration++){
                gpuCheck( hipMemcpy(d_A, h_A, bytes, hipMemcpyHostToDevice) );
            }        
        }
        else{
            printf("Error - unknown direction\n");
            exit(1);
        }

        gpuCheck( hipDeviceSynchronize() );
        gpuCheck( hipEventRecord(start, NULL) );

        /* Timed loop */
        if( strcmp(direction, "H2D") == 0){

            for(int iteration=0; iteration<loop_count; iteration++){
                gpuCheck( hipMemcpy(d_A, h_A, bytes, hipMemcpyHostToDevice) );
            }
        }
        else if( strcmp(direction, "D2H") == 0){

            for(int iteration=0; iteration<loop_count; iteration++){
                gpuCheck( hipMemcpy(d_A, h_A, bytes, hipMemcpyHostToDevice) );
            }
        }
        else{
            printf("Error - unknown direction\n");
            exit(1);
        }

        gpuCheck( hipEventRecord(stop, NULL) );
        gpuCheck( hipEventSynchronize(stop) );
        gpuCheck( hipEventElapsedTime(&milliseconds, start, stop) );

        double bandwidth = ( 1000.0 * (double)loop_count * (double)bytes ) / ( (double)milliseconds * 1000.0 * 1000.0 * 1000.0);
        double bytes_mb  = (double)bytes / (1024.0 * 1024.0);

        printf("Buffer Size (MiB): %14.9f, Time (ms): %14.9f, Bandwidth (GB/s): %14.9f\n", bytes_mb, milliseconds, bandwidth);
    }
}

/* --------------------------------------------------
Main program
-------------------------------------------------- */
int main(int argc, char *argv[])
{
    printf("----- H2D -----\n");
    host_device_transfer("H2D");

    printf("----- D2H -----\n");
    host_device_transfer("D2H");

    printf("\n__SUCCESS__\n");

    return 0;
}
